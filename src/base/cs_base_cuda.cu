#include "hip/hip_runtime.h"
/*============================================================================
 * Definitions, global variables, and base functions for CUDA
 *============================================================================*/

/*
  This file is part of code_saturne, a general-purpose CFD tool.

  Copyright (C) 1998-2025 EDF S.A.

  This program is free software; you can redistribute it and/or modify it under
  the terms of the GNU General Public License as published by the Free Software
  Foundation; either version 2 of the License, or (at your option) any later
  version.

  This program is distributed in the hope that it will be useful, but WITHOUT
  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
  FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more
  details.

  You should have received a copy of the GNU General Public License along with
  this program; if not, write to the Free Software Foundation, Inc., 51 Franklin
  Street, Fifth Floor, Boston, MA 02110-1301, USA.
*/

/*----------------------------------------------------------------------------*/

#include "base/cs_defs.h"

/*----------------------------------------------------------------------------
 * Standard C and C++ library headers
 *----------------------------------------------------------------------------*/

/*----------------------------------------------------------------------------
 * Local headers
 *----------------------------------------------------------------------------*/

#include "assert.h"
#include "bft/bft_error.h"
#include "bft/bft_printf.h"

#include "base/cs_base.h"
#include "base/cs_log.h"
#include "base/cs_mem.h"
#include "base/cs_mem_cuda_priv.h"

/*----------------------------------------------------------------------------
 *  Header for the current file
 *----------------------------------------------------------------------------*/

#include "base/cs_base_cuda.h"

/*----------------------------------------------------------------------------*/

BEGIN_C_DECLS

/*! \cond DOXYGEN_SHOULD_SKIP_THIS */

/*============================================================================
 * Local Macro Definitions
 *============================================================================*/

/*============================================================================
 * Local Type Definitions
 *============================================================================*/

/*============================================================================
 *  Global variables
 *============================================================================*/

/* Keep track of active device id; usually queried dynamically, but
   saving the value in this variable can be useful when debugging */

int  cs_glob_cuda_device_id = -1;

/* Other device parameters */

int  cs_glob_cuda_max_threads_per_block = -1;
int  cs_glob_cuda_max_block_size = -1;
int  cs_glob_cuda_max_blocks = -1;
int  cs_glob_cuda_n_mp = -1;

/* Stream pool */

static int            _cs_glob_cuda_n_streams = -1;
static hipStream_t  *_cs_glob_cuda_streams = nullptr;

/* Reduce buffers associated with streams in pool */

static unsigned *_r_elt_size = nullptr;
static unsigned *_r_grid_size = nullptr;
static void  **_r_reduce = nullptr;
static void  **_r_grid = nullptr;

static hipStream_t _cs_glob_stream_pf = 0;

/* Allow graphs for kernel launches ? May interfere with profiling (nsys),
   so can be deactivated. */

bool cs_glob_cuda_allow_graph = false;

// Shared memory size ber block (based on know GPUs, queried later).
size_t cs_glob_cuda_shared_mem_per_block = 0x19000;

/*============================================================================
 * Private function definitions
 *============================================================================*/

/*----------------------------------------------------------------------------*/
/*!
 * \brief Destroy stream pool at exit.
 */
/*----------------------------------------------------------------------------*/

static void
finalize_streams_(void)
{
  cs_mem_cuda_set_prefetch_stream(0);
  hipStreamDestroy(_cs_glob_stream_pf);

  for (int i = 0; i < _cs_glob_cuda_n_streams; i++) {
    hipStreamDestroy(_cs_glob_cuda_streams[i]);
    CS_FREE(_r_reduce[i]);
    CS_FREE(_r_grid[i]);
  }

  CS_FREE(_cs_glob_cuda_streams);

  CS_FREE(_r_elt_size);
  CS_FREE(_r_grid_size);
  CS_FREE(_r_reduce);
  CS_FREE(_r_grid);

  _cs_glob_cuda_n_streams = 0;
}

/*============================================================================
 * Semi-private function prototypes
 *
 * The following functions are intended to be used by the common
 * host-device memory management functions from cs_base_accel.c, and
 * not directly by the user.
 *============================================================================*/

/*----------------------------------------------------------------------------*/
/*!
 * \brief Copy data from host to device.
 *
 * This is simply a wrapper over hipMemcpy.
 *
 * A safety check is added.
 *
 * \param [out]  dst   pointer to destination data
 * \param [in]   src   pointer to source data
 * \param [in]   size  size of data to copy
 */
/*----------------------------------------------------------------------------*/

void
cs_cuda_copy_h2d(void        *dst,
                 const void  *src,
                 size_t       size)
{
  CS_CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Copy data from host to device, possibly returning on the host
 *        before the copy is finished.
 *
 * This is simply a wrapper over hipMemcpyAsync.
 *
 * A safety check is added.
 *
 * \param [out]  dst   pointer to destination data
 * \param [in]   src   pointer to source data
 * \param [in]   size  size of data to copy
 *
 * \returns pointer to allocated memory.
 */
/*----------------------------------------------------------------------------*/

void
cs_cuda_copy_h2d_async(void        *dst,
                       const void  *src,
                       size_t       size)
{
  CS_CUDA_CHECK(hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice));
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Copy data from device to host.
 *
 * This is simply a wrapper over hipMemcpy.
 *
 * A safety check is added.
 *
 * \param [out]  dst   pointer to destination data
 * \param [in]   src   pointer to source data
 * \param [in]   size  size of data to copy
 *
 * \returns pointer to allocated memory.
 */
/*----------------------------------------------------------------------------*/

void
cs_cuda_copy_d2h(void        *dst,
                 const void  *src,
                 size_t       size)
{
  CS_CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost));
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Copy data from host to device.
 *
 * This is simply a wrapper over hipMemcpy.
 *
 * A safety check is added.
 *
 * \param [out]  dst   pointer to destination data
 * \param [in]   src   pointer to source data
 * \param [in]   size  size of data to copy
 *
 * \returns pointer to allocated memory.
 */
/*----------------------------------------------------------------------------*/

void
cs_cuda_copy_d2h_async(void        *dst,
                       const void  *src,
                       size_t       size)
{
  CS_CUDA_CHECK(hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost));
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Copy data from device to device.
 *
 * This is simply a wrapper over hipMemcpy.
 *
 * A safety check is added.
 *
 * \param [out]  dst   pointer to destination data
 * \param [in]   src   pointer to source data
 * \param [in]   size  size of data to copy
 */
/*----------------------------------------------------------------------------*/

void
cs_cuda_copy_d2d(void        *dst,
                 const void  *src,
                 size_t       size)
{
  CS_CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice));
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Get host pointer for a managed or device pointer.
 *
 * This function can be called with a pointer inside an allocated block of
 * memory, so is not retricted to values returned by CS_ALLOC_HD.
 *
 * This makes it possible to check whether a pointer to an array inside
 * a larger array is shared or accessible from the device only
 * (for example when grouping allocations).
 *
 * \param [in]   ptr   pointer to device data
 *
 * \return  pointer to host data if shared or mapped at the CUDA level,
 *          NULL otherwise.
 */
/*----------------------------------------------------------------------------*/

void *
cs_cuda_get_host_ptr(const void  *ptr)
{
  hipPointerAttribute_t attributes;

  void *host_ptr = nullptr;
  int retcode = hipPointerGetAttributes(&attributes, ptr);

  if (retcode == hipSuccess) {
    if (ptr != attributes.devicePointer)
      bft_error(__FILE__, __LINE__, 0,
                _("%s: %p does not seem to be a managed or device pointer."),
                __func__, ptr);

    host_ptr = attributes.hostPointer;
  }

  return host_ptr;
}

/*! (DOXYGEN_SHOULD_SKIP_THIS) \endcond */

END_C_DECLS

/*============================================================================
 * Public function definitions
 *============================================================================*/

#ifdef __HIPCC__

/*----------------------------------------------------------------------------*/
/*!
 * \brief Return stream handle from stream pool.
 *
 * If the requested stream id is higher than the current number of streams,
 * one or more new streams will be created, so that size of the stream pool
 * matches at least stream_id+1.
 *
 * By default, the first stream (with id 0) will be used for most operations,
 * while stream id 1 will be used for operations which can be done
 * concurrently, such as memory prefetching.
 *
 * Additional streams can be used for independent tasks, though opportunities
 * for this are limited in the current code (this would probably also require
 * associating different MPI communicators with each task).
 *
 * \param [in]  stream_id  id or requested stream
 *
 * \returns handle to requested stream
 */
/*----------------------------------------------------------------------------*/

hipStream_t
cs_cuda_get_stream(int  stream_id)
{
  if (stream_id >= 0 && stream_id < _cs_glob_cuda_n_streams)
    return _cs_glob_cuda_streams[stream_id];
  else if (stream_id < 0)
    return nullptr;

  if (_cs_glob_cuda_n_streams < 0) {
    cs_base_at_finalize(finalize_streams_);
    _cs_glob_cuda_n_streams = 0;
  }

  CS_REALLOC(_cs_glob_cuda_streams, stream_id+1, hipStream_t);
  CS_REALLOC(_r_elt_size, stream_id+1, unsigned);
  CS_REALLOC(_r_grid_size, stream_id+1, unsigned);
  CS_REALLOC(_r_reduce, stream_id+1, void *);
  CS_REALLOC(_r_grid, stream_id+1, void *);

  for (int i = _cs_glob_cuda_n_streams; i < stream_id+1; i++) {
    hipStreamCreate(&_cs_glob_cuda_streams[i]);
    _r_elt_size[i] = 0;
    _r_grid_size[i] = 0;
    _r_reduce[i] = nullptr;
    _r_grid[i] = nullptr;
  }

  _cs_glob_cuda_n_streams = stream_id+1;

  return _cs_glob_cuda_streams[stream_id];
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Return stream handle used for prefetching.
 *
 * By default, a single stream is created specifically for prefetching.
 *
 * \returns handle to prefetching stream
 */
/*----------------------------------------------------------------------------*/

hipStream_t
cs_cuda_get_stream_prefetch(void)
{
  return _cs_glob_stream_pf;
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Return stream id in stream pool matching a given CUDA stream.
 *
 * If the stream is not presnet in the stream pool, return -1.
 *
 * \param [in]  handle to given streams
 *
 * \returns if of stream in pool, or -1.
 */
/*----------------------------------------------------------------------------*/

int
cs_cuda_get_stream_id(hipStream_t  stream)
{
  for (int i = 0; i < _cs_glob_cuda_n_streams; i++)
    if (stream == _cs_glob_cuda_streams[i])
      return i;

  return -1;
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Return pointers to reduction buffers needed for 2-stage reductions.
 *
 * These buffers are used internally by CUDA 2-stage operations, and are
 * allocated and resized updon demand.
 *
 * \param[in]   stream_id   stream id in pool
 * \param[in]   n_elts      size of arrays
 * \param[in]   n_elts      size of arrays
 * \param[in]   elt_size    size of element or structure simultaneously reduced
 * \param[in]   grid_size   associated grid size
 * \param[out]  r_grid      first stage reduce buffer
 * \param[out]  r_reduce    second stage (final result) reduce buffer
 */
/*----------------------------------------------------------------------------*/

void
cs_cuda_get_2_stage_reduce_buffers(int            stream_id,
                                   cs_lnum_t      n_elts,
                                   size_t         elt_size,
                                   unsigned int   grid_size,
                                   void*         &r_grid,
                                   void*         &r_reduce)
{
  assert(stream_id > -1 && stream_id < _cs_glob_cuda_n_streams);

  unsigned int t_grid_size = grid_size * elt_size;

  if (_r_elt_size[stream_id] < elt_size) {
    _r_elt_size[stream_id] = elt_size;
    CS_FREE_HD(_r_reduce[stream_id]);
    unsigned char *b_ptr;
    CS_MALLOC_HD(b_ptr, elt_size, unsigned char, CS_ALLOC_HOST_DEVICE_SHARED);
    _r_reduce[stream_id] = b_ptr;
  }

  if (_r_grid_size[stream_id] < t_grid_size) {
    _r_grid_size[stream_id] = t_grid_size;
    CS_FREE(_r_grid[stream_id]);
    unsigned char *b_ptr;
    CS_MALLOC_HD(b_ptr, _r_grid_size[stream_id], unsigned char, CS_ALLOC_DEVICE);
    _r_grid[stream_id] = b_ptr;
  }

  r_grid = _r_grid[stream_id];
  r_reduce = _r_reduce[stream_id];
}

#endif /* defined(__HIPCC__) */

/*----------------------------------------------------------------------------*/
/*!
 * \brief  Log information on available CUDA devices.
 *
 * \param[in]  log_id  id of log file in which to print information
 */
/*----------------------------------------------------------------------------*/

extern "C" void
cs_base_cuda_device_info(cs_log_t  log_id)
{
  int n_devices = 0;

  hipError_t retval = hipGetDeviceCount(&n_devices);

  if (retval == hipErrorNoDevice)
    cs_log_printf(log_id,
                  _("  CUDA device:         none available\n"));
  else if (retval)
    cs_log_printf(log_id,
                  _("  CUDA device:         %s\n"),
                  hipGetErrorString(retval));

  char buffer[256] = "";

  for (int i = 0; i < n_devices; i++) {
    struct hipDeviceProp_t prop;
    CS_CUDA_CHECK(hipGetDeviceProperties(&prop, i));
    unsigned long long mem = prop.totalGlobalMem / 1000000;

    cs_glob_cuda_shared_mem_per_block = prop.sharedMemPerBlock;

    cs_log_printf
      (log_id,
       _("  CUDA device %d:       %s\n"),
       i, prop.name);

    if (strncmp(prop.name, buffer, 255) != 0) {
      cs_log_printf
        (log_id,
         _("                       Compute capability: %d.%d\n"
           "                       Memory: %llu %s\n"
           "                       Multiprocessors: %d\n"
           "                       Integrated: %d\n"
           "                       Unified addressing: %d\n"),
         prop.major, prop.minor,
         mem, _("MB"),
         prop.multiProcessorCount,
         prop.integrated,
         prop.unifiedAddressing);

#if (CUDART_VERSION >= 11000)
      cs_log_printf
        (log_id,
         _("                       Use host's page tables: %d\n"),
         prop.pageableMemoryAccessUsesHostPageTables);
#endif
    }

    strncpy(buffer, prop.name, 255);
    buffer[255] = '\0';
  }
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief  Log information on available CUDA version.
 *
 * \param[in]  log_id  id of log file in which to print information
 */
/*----------------------------------------------------------------------------*/

extern "C" void
cs_base_cuda_version_info(cs_log_t  log_id)
{
  int runtime_version = -1, driver_version = -1;

  if (hipDriverGetVersion(&driver_version) == hipSuccess)
    cs_log_printf(log_id,
                  "  %s%d\n", _("CUDA driver:         "), driver_version);
  if (hipRuntimeGetVersion(&runtime_version) == hipSuccess)
    cs_log_printf(log_id,
                  "  %s%d\n", _("CUDA runtime:        "), runtime_version);
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief  Log information on CUDA compiler.
 *
 * \param[in]  log_id  id of log file in which to print information
 */
/*----------------------------------------------------------------------------*/

extern "C" void
cs_base_cuda_compiler_info(cs_log_t  log_id)
{
  cs_log_printf(log_id,
                "    %s%d.%d.%d\n", _("CUDA compiler:     "),
                __CUDACC_VER_MAJOR__,
                __CUDACC_VER_MINOR__,
                __CUDACC_VER_BUILD__);
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Set CUDA device based on MPI rank and number of devices.
 *
 * \param[in]  comm            associated MPI communicator
 * \param[in]  ranks_per_node  number of ranks per node (min and max)
 *
 * \return  selected device id, or -1 if no usable device is available
 */
/*----------------------------------------------------------------------------*/

extern "C" int
cs_base_cuda_select_default_device(void)
{
  int device_id = 0, n_devices = 0;

  hipError_t ret_code = hipGetDeviceCount(&n_devices);

  if (ret_code == hipErrorNoDevice)
    return -1;

  if (hipSuccess != ret_code) {
    cs_base_warn(__FILE__, __LINE__);
    bft_printf("[CUDA error] %d: %s\n  running: %s\n  in: %s\n",
               ret_code, ::hipGetErrorString(ret_code),
               "hipGetDeviceCount", __func__);
    return -1;
  }

  if (cs_glob_rank_id > -1 && n_devices > 1) {

    device_id = cs_glob_node_rank_id*n_devices / cs_glob_node_n_ranks;

    assert(device_id > -1 && device_id < n_devices);

  }

  ret_code = hipSetDevice(device_id);

  if (hipSuccess != ret_code) {
    cs_base_warn(__FILE__, __LINE__);
    bft_printf("[CUDA error] %d: %s\n  running: %s\n  in: %s\n",
               ret_code, ::hipGetErrorString(ret_code),
               "hipSetDevice", __func__);
    return -1;
  }

  cs_glob_cuda_device_id = device_id;

  cs_alloc_mode = CS_ALLOC_HOST_DEVICE_SHARED;
  cs_alloc_mode_read_mostly = CS_ALLOC_HOST_DEVICE_SHARED;

  /* Also query some device properties */

  struct hipDeviceProp_t prop;
  CS_CUDA_CHECK(hipGetDeviceProperties(&prop, device_id));
  cs_glob_cuda_max_threads_per_block = prop.maxThreadsPerBlock;
  cs_glob_cuda_max_block_size = prop.maxThreadsPerMultiProcessor;
  cs_glob_cuda_max_blocks
    =   prop.multiProcessorCount
      * (prop.maxThreadsPerMultiProcessor / prop.maxThreadsPerBlock);
  cs_glob_cuda_n_mp = prop.multiProcessorCount;

  /* Create default stream for prefetching */
  if (_cs_glob_stream_pf == 0) {
    hipStreamCreate(&_cs_glob_stream_pf);
    cs_mem_cuda_set_prefetch_stream(_cs_glob_stream_pf);
  }

  /* Finally, determine whether we may use graphs for some kernel launches. */

  const char s[] = "CS_CUDA_ALLOW_GRAPH";
  if (getenv(s) != nullptr) {
    int i = atoi(getenv(s));
    cs_glob_cuda_allow_graph = (i <= 0) ? false : true;
  }

  return device_id;
}

/*----------------------------------------------------------------------------*/
/*!
 * \brief Return currently selected CUDA devices.
 *
 * \return  selected device id, or -1 if no usable device is available
 */
/*----------------------------------------------------------------------------*/

extern "C" int
cs_base_cuda_get_device(void)
{
  int device_id = -1, n_devices = 0;

  hipError_t ret_code = hipGetDeviceCount(&n_devices);

  if (hipSuccess == ret_code)
    ret_code = hipGetDevice(&device_id);

  if (hipSuccess != ret_code)
    device_id = -1;

  return device_id;
}

/*----------------------------------------------------------------------------*/
